#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void Vec_add(float x[], float y[], float z[], int n) {
   int i = blockDim.x * blockIdx.x + threadIdx.x;
   if (i < n){
     z[i] = x[i] + y[i];
   }
 }

int main(int argc, char* argv[]) {
   int n, i;
   float *h_x, *h_y, *h_z;
   float *d_x, *d_y, *d_z;
   int threads_per_block;
   int block_count;
   size_t size;
   if (argc != 2) {
      fprintf(stderr, "usage: %s <vector order>\n", argv[0]);
      exit(0);
   }
   n = strtol(argv[1], NULL, 10);
   size = n*sizeof(float);

   h_x = (float*) malloc(size);
   h_y = (float*) malloc(size);
   h_z = (float*) malloc(size);

   for (i = 0; i < n; i++) {
      h_x[i] = i+1;
      h_y[i] = n-i;
   }

   hipMalloc(&d_x, size);
   hipMalloc(&d_y, size);
   hipMalloc(&d_z, size);
   hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
   hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

   threads_per_block = 256;
   block_count = (n + threads_per_block - 1)/threads_per_block;
   Vec_add<<<block_count, threads_per_block>>>(d_x, d_y, d_z, n);
   hipDeviceSynchronize();
   hipMemcpy(h_z, d_z, size, hipMemcpyDeviceToHost);

   printf("The sum is: \n");
   for (i = 0; i < n; i++)
      printf("%.1f ", h_z[i]);
   printf("\n");

   hipFree(d_x);
   hipFree(d_y);
   hipFree(d_z);
   free(h_x);
   free(h_y);
   free(h_z);
   return 0;
}
