#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <math.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>

//number of channels i.e. R G B
#define CHANNELS 3
#define BLUR_SIZE 6

//Cuda kernel for blurring an RGB image
__global__
void add_blur(unsigned char *rgb, unsigned char *blur_rgb, int rows, int cols)
{
	int col = threadIdx.x + blockIdx.x * blockDim.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;

	//Compute for only those threads which map directly to
	//image grid
	if (col < cols && row < rows)
	{
        int r_pixel_val = 0;
        int g_pixel_val = 0;
        int b_pixel_val = 0;
        int pixels = 0;
        int blur_row = 0;
        int blur_col = 0;
        for (blur_row = -BLUR_SIZE; blur_row < BLUR_SIZE + 1; ++blur_row)
        {
            for (blur_col = -BLUR_SIZE; blur_col < BLUR_SIZE + 1; ++blur_col)
            {
                int cur_row = row + blur_row;
                int cur_col = col + blur_col;

                if (cur_row > -1 && cur_row < rows && cur_col > -1 && cur_col < cols)
                {

                    int offset = (cur_row * cols + cur_col) * CHANNELS;
					//weight sum for all 3 channels separately
                    r_pixel_val += rgb[offset + 0];
                    g_pixel_val += rgb[offset + 1];
                    b_pixel_val += rgb[offset + 2];
                    pixels++;
                }
            }
        }

        int blur_offset = (row * cols + col) * CHANNELS;

        blur_rgb[blur_offset + 0] = (unsigned char) (r_pixel_val / pixels);
        blur_rgb[blur_offset + 1] = (unsigned char) (g_pixel_val / pixels);
        blur_rgb[blur_offset + 2] = (unsigned char) (b_pixel_val / pixels);
    }
}

size_t loadImageFile(unsigned char *blur_image, const std::string &input_file, int *rows, int *cols );

void outputImage(const std::string &output_file, unsigned char *blur_image, int rows, int cols);

unsigned char *h_rgb_image; //store image's rbg data

int main(int argc, char **argv)
{
	std::string input_file;
	std::string output_file;

	//Check for the input file and output file names
	switch(argc) {
		case 3:
			input_file = std::string(argv[1]);
			output_file = std::string(argv[2]);
            break;
		default:
			std::cerr << "Usage: <executable> input_file output_file";
			exit(1);
	}

	unsigned char *d_rgb_image; //array for storing rgb data on device
	unsigned char *h_blur_image, *d_blur_image; //host and device's blur image data array pointers
	int rows; //number of rows of pixels
	int cols; //number of columns of pixels

	//load image into an array and retrieve number of pixels
	const size_t total_pixels = loadImageFile(h_blur_image, input_file, &rows, &cols);

	//allocate memory of host's blur image data array
	h_blur_image = (unsigned char *)malloc(sizeof(unsigned char*) * total_pixels * CHANNELS);

	//allocate and initialize memory on device
	hipMalloc(&d_rgb_image, sizeof(unsigned char) * total_pixels * CHANNELS);
	hipMalloc(&d_blur_image, sizeof(unsigned char) * total_pixels * CHANNELS);
	hipMemset(d_blur_image, 0, sizeof(unsigned char) * total_pixels * CHANNELS);

	//copy host rgb data array to device rgb data array
	hipMemcpy(d_rgb_image, h_rgb_image, sizeof(unsigned char) * total_pixels * CHANNELS, hipMemcpyHostToDevice);

	//define block and grid dimensions
	const dim3 dimGrid((int)ceil((cols)/16), (int)ceil((rows)/16));
	const dim3 dimBlock(16, 16);

	//execute cuda kernel
	add_blur<<<dimGrid, dimBlock>>>(d_rgb_image, d_blur_image, rows, cols);

	//copy computed gray data array from device to host
	hipMemcpy(h_blur_image, d_blur_image, sizeof(unsigned char) * total_pixels * CHANNELS, hipMemcpyDeviceToHost);

	//output the blurred image
	outputImage(output_file, h_blur_image, rows, cols);
	hipFree(d_rgb_image);
	hipFree(d_blur_image);
	return 0;
}

//function for loading an image into rgb format unsigned char array
size_t loadImageFile(unsigned char *blur_image, const std::string &input_file, int *rows, int *cols)
{
	cv::Mat img_data; //opencv Mat object

	//read image data into img_data Mat object
	img_data = cv::imread(input_file.c_str(), CV_LOAD_IMAGE_COLOR);
	if (img_data.empty())
	{
		std::cerr << "Unable to laod image file: " << input_file << std::endl;
	}

	*rows = img_data.rows;
	*cols = img_data.cols;

	//allocate memory for host rgb data array
	h_rgb_image = (unsigned char*) malloc(*rows * *cols * sizeof(unsigned char) * CHANNELS);
	unsigned char* rgb_image = (unsigned char*)img_data.data;

	//populate host's rgb data array
	int x = 0;
	for (x = 0; x < *rows * *cols * CHANNELS; x++)
	{
		h_rgb_image[x] = rgb_image[x];
	}

	size_t num_of_pixels = img_data.rows * img_data.cols;

	return num_of_pixels;
}

//function for writing blurred image data array to the image file
void outputImage(const std::string& output_file, unsigned char* blur_image, int rows, int cols)
{
	//serialize gray data array into opencv's Mat object
	cv::Mat blurData(rows, cols, CV_8UC3,(void *) blur_image);
	//write Mat object to file
	cv::imwrite(output_file.c_str(), blurData);
}
